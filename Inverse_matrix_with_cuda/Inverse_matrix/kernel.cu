#include "hip/hip_runtime.h"
//#include "hip/hip_runtime.h"
#include ""
#include <hiprand/hiprand_kernel.h>//��������� ������
#include <hip/hip_runtime.h>
#include <iostream>
#include <cassert>
#include <cmath>
#include <time.h>

using namespace std;
//__global__ - ���������� CPU, �� ����������� GPU\
__device__ - ����������� �� GPU\
__host__ - ����������� �� CPU

//����������, � ������� ����� ��������� ��������� ������� max_in_column()\
����� ���� ������� ������ �� GPU
__device__ int max_element_index;

//����� ������������� �������� � �������, ����������� �� GPU
__global__ void max_in_column(double* matrix, int matrix_size, int x)
{
	int max_index = x;
	double max_value = fabs(matrix[x * matrix_size + x]);
	double current_value;
	for (int i = x + 1; i < matrix_size; i++)
	{
		//���������� �� ������ ������ ���
		current_value = fabs(matrix[i * matrix_size + x]);//fabs - ������ ������
		if (current_value > max_value)
		{
			max_index = i;
			max_value = current_value;
		}
	}
	max_element_index = max_index;//��������� ������������ ������ �������
}

__global__ void swap_lines(double* matrix, double* identity, int matrix_size, int row) {
	//���� ��������� ������ � ��� �� ����� �����, �� ������ ������ �� �����
	if (row == max_element_index)
		return;
	//������ �����(���������� ������� � �����)*����� �����+����� ������
	int idx = blockIdx.x * blockDim.x + threadIdx.x;
	if (idx >= matrix_size) return;//���������, ��� �� ������� �� ������� �������
	double temp;
	temp = matrix[row * matrix_size + idx];//��������� ������� ������
	//������ �� ����� ����� �������� ��������������� ��� ������� ������ ������
	matrix[row * matrix_size + idx] = matrix[max_element_index * matrix_size + idx];
	//������ ����������� ������� �� ��������������� ��� ����� � ������ ������
	matrix[max_element_index * matrix_size + idx] = temp;

	//������ ����������� ��������, �� � ������ ��������
	temp = identity[row * matrix_size + idx];
	identity[row * matrix_size + idx] = identity[max_element_index * matrix_size + idx];
	identity[max_element_index * matrix_size + idx] = temp;
}

__global__ void make_zero_below(double* matrix, double* identity, int matrix_size, int x)
{
	//������ ����� ������
	int idx = threadIdx.x;
	int i, j;
	
	//���� ��� ������, ��� ������� ���� �������, �� ������ �� ��������, ���� �� ���� ������ �������� ������
	if (idx * matrix_size == x * matrix_size) return;
	
	//������� �����������
	double 	coeff = matrix[idx * matrix_size + x] / matrix[x * matrix_size + x];

	//�������� ������ x � �������� matrix[x][x] �� ����� ������ � �� ������� �������� ��������\
	��������������� ������� ������ x, ���������� �� coeff
	for (i = x;i < matrix_size;i++)
		matrix[idx * matrix_size + i] -= coeff* matrix[x * matrix_size + i];
	
	//������ ���� ����� ��� ������ ����� ����������� �������
	for (;i < 2*matrix_size;i++)
		identity[idx * matrix_size + i] -= coeff * identity[x * matrix_size + i];


}

void print_both(double* matrix, double* identity_matrix, int matrix_size);
void print(double* matrix, int n);

__host__ int main()
{
	//���� ������� �������
	cout << "Enter the size of the matrix: ";
	int matrix_size;
	cin>> matrix_size;

	//���������� �������
	//��������� ������ ��� ������ �� CPU
	double* matrix = new double[matrix_size * matrix_size];
	assert(matrix != 0);
	double* identity_matrix = new double[matrix_size * matrix_size];
	assert(identity_matrix != 0);

	//��������� ������� ���������� �������
	srand(time(NULL));
	for (int i = 0; i < matrix_size * matrix_size; i++)
		matrix[i] = 1 + rand() % 10;

	//��������� ��������� �������
	for (int i = 0; i < matrix_size; i++)
		for (int j = 0; j < matrix_size; j++)
			identity_matrix[i * matrix_size + j] = (i == j ? 1.0 : 0.0);

	cout<< "Before:"<< endl;
	print_both(matrix, identity_matrix, matrix_size);

	//��������� ������ ��� ������ �� GPU
	double* dev_matrix;
	double* dev_identity;
	hipMalloc((void**)&dev_matrix, sizeof(double) * matrix_size * matrix_size);
	hipMalloc((void**)&dev_identity, sizeof(double) * matrix_size * matrix_size);

	//�������� � host �� device
	hipMemcpy(dev_matrix, matrix, sizeof(double) * matrix_size * matrix_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_identity, identity_matrix, sizeof(double) * matrix_size * matrix_size, hipMemcpyHostToDevice);

	//�������������� ���������� ��� ������ ������� ����� �� GPU
	float recording;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//�� ������ ����� ��� ������ � ����� ���������� � ���������������\
	� ������� ������������ �������������� ������� ������� ��� ������� ����������
	//int i = 0;
	for (int i = 0; i < matrix_size; i++)
	{
	// �������� ������ �� ������� � ���������� ������� ��������� � ��������
	// �������� ��� ���
	
	//���� ������������ ������� �������
	max_in_column<< <1, 1 >> > (dev_matrix, matrix_size, i);
	
	//������ i-�� ������ �� ������� � ������������ ��������� � i-�� ������� �������
	swap_lines << <1, matrix_size>> >(dev_matrix, dev_identity, matrix_size, i);

	//�������� �������� ��� i-�� � i-�� �������
	make_zero_below << < 1, matrix_size-i >> > (dev_matrix, dev_identity, matrix_size, i);
	
	hipMemcpy(matrix, dev_matrix, sizeof(double) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
	hipMemcpy(identity_matrix, dev_identity, sizeof(double) * matrix_size * matrix_size, hipMemcpyDeviceToHost);

	cout << "After:" << endl;
	print_both(matrix, identity_matrix, matrix_size);
	//���-��
	/*nullify_below << <BLOCKS_1D, THREADS_1D >> > (dev_matrix, n, i);*/
	}
	hipDeviceSynchronize();//������������� �������

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&recording, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//�������� ��������� �� ������ GPU �� CPU
	hipMemcpy(matrix, dev_matrix, sizeof(double) * matrix_size * matrix_size, hipMemcpyDeviceToHost);
	hipMemcpy(identity_matrix, dev_identity, sizeof(double) * matrix_size * matrix_size, hipMemcpyDeviceToHost);

	cout << "After:" << endl;
	print_both(matrix, identity_matrix, matrix_size);

	//����� �������� �������
	cout << "Invers matrix:" << endl;
	print(identity_matrix, matrix_size);

	cout << "Time of work: " << fixed << recording << endl;

	//����������� ������ device
	hipFree(dev_matrix);
	hipFree(dev_identity);

	//����������� ������ host
	delete[] matrix;
	delete[] identity_matrix;
	return 0;
}

__host__ void print_both(double* matrix, double* identity_matrix, int matrix_size)
{
	for (int i = 0; i < matrix_size; i++)
	{
		int flag = 1;
		for (int j = 0; j < 2 * matrix_size; j++)
			if (j < matrix_size)
				cout <<fixed << matrix[i * matrix_size + j] << "\t";
			else
			{
				if (flag)
				{
					cout << "|\t";
					flag--;
				}
				cout << fixed << identity_matrix[i * matrix_size + j - matrix_size] << "\t";
			}
		cout << endl;
	}
	cout << endl;
}

__host__ void print(double* matrix, int n)
{
	cout << scientific;
	for (int i = 0; i < n; i++)
	{
		for (int j = 0; j < n; j++)
			cout << fixed << matrix[i * n + j] << "\t";
		cout << endl;
	}
}
